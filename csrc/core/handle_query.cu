#include "hip/hip_runtime.h"
#include "hetu_gpu_table.h"
#include "common/hip/hip_runtime_api.h"
#include <hipcub/hipcub.hpp>

namespace hetuCTR {

__global__ void computeReturnOutdated(HetuGPUTable *tbl, size_t len) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len) {
    version_t local_version = tbl->d_query_version_[1][id];
    index_t embedding_idx = tbl->d_query_idx_[1][id];
    auto iter = tbl->table_->find(embedding_idx);

    assert(tbl->d_root_[embedding_idx] == tbl->rank_);
    assert(iter != tbl->table_->end());

    version_t global_version = tbl->d_version_[iter->second];
    if (local_version == kInvalidVersion || local_version + tbl->pull_bound_ <= global_version)
      tbl->d_return_outdated_[0][id] = 1;
    else tbl->d_return_outdated_[0][id] = 0;
  }
}

__global__ void write_return_value_kernel(HetuGPUTable *tbl) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  size_t len = *(tbl->d_shape_);
  if (id < len) {
    version_t local_version = tbl->d_query_version_[1][id];
    index_t embedding_idx = tbl->d_update_prefix_[id];
    auto iter = tbl->table_->find(embedding_idx);

    assert(tbl->d_root_[embedding_idx] == tbl->rank_);
    assert(iter != tbl->table_->end());
    index_t offset = iter->second;

    version_t global_version = tbl->d_version_[offset];
    tbl->d_return_version_[0][id] = global_version;
    for (int i = 0; i < tbl->kEmbeddingWidth; i++)
      tbl->d_return_val_[0][tbl->kEmbeddingWidth * id + i] = tbl->d_embedding_[tbl->kEmbeddingWidth * offset + i];
  }
}

void HetuGPUTable::handleQuery() {
  INFO(all2all_received_, " received embedding index to handle.");
  computeReturnOutdated<<<DIM_GRID(all2all_received_), DIM_BLOCK, 0, stream_main_>>>(d_this, all2all_received_);

  all2allReturnOutdated();

  checkCudaErrors(hipcub::DeviceScan::ExclusiveSum(d_temp_, temp_bytes_,
    cur_batch_.u_shape_exchanged, cur_batch_.u_shape_exchanged, nrank_ + 1, stream_main_));

  checkCudaErrors(hipcub::DeviceSegmentedReduce::Sum(d_temp_, temp_bytes_,
    d_return_outdated_[0], cur_batch_.u_shape, nrank_,
    cur_batch_.u_shape_exchanged, cur_batch_.u_shape_exchanged + 1, stream_main_));

  // exchange return value shape and copy them to host
  all2allExchangeShape(cur_batch_.u_shape, cur_batch_.u_shape_exchanged);
  checkCudaErrors(hipMemcpyAsync(cur_batch_.h_shape, cur_batch_.u_shape,
    sizeof(size_t) * (nrank_ + 1), hipMemcpyDeviceToHost, stream_main_));
  checkCudaErrors(hipMemcpyAsync(cur_batch_.h_shape_exchanged, cur_batch_.u_shape_exchanged,
    sizeof(size_t) * (nrank_ + 1), hipMemcpyDeviceToHost, stream_main_));

  // select index that requires update into d_update_prefix_
  // total number stored in d_shape_
  checkCudaErrors(hipcub::DeviceSelect::Flagged(d_temp_, temp_bytes_,
    d_query_idx_[1], d_return_outdated_[0], d_update_prefix_, d_shape_, all2all_received_, stream_main_));

  write_return_value_kernel<<<DIM_GRID(all2all_received_), DIM_BLOCK, 0, stream_main_>>>(d_this);

  checkCudaErrors(hipStreamSynchronize(stream_main_));

  all2allReturnValue();
}

__global__ void table_update_remote_kernel(HetuGPUTable *tbl, size_t start, size_t len) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len) {
    size_t width = tbl->kEmbeddingWidth;
    id += start;
    index_t embedding_idx = tbl->d_query_gradient_idx_[1][id];
    auto iter = tbl->table_->find(embedding_idx);

    assert(tbl->d_root_[embedding_idx] == tbl->rank_);
    assert(iter != tbl->table_->end());
    index_t offset = iter->second;

    tbl->d_version_[offset] += tbl->d_query_updates_[1][id];
    for (int i = 0; i < tbl->kEmbeddingWidth; i++)
      tbl->d_embedding_[offset * width + i] += tbl->d_query_val_[1][id * width + i];
  }
}

void HetuGPUTable::handleGradient() {
  size_t offset = 0;
  for (int i = 0 ; i < nrank_; i++) {
    size_t shape = prev_batch_.h_shape_exchanged[i];
    table_update_remote_kernel<<<DIM_GRID(shape), DIM_BLOCK, 0, stream_main_>>>(d_this, offset, shape);
    offset += shape;
  }
}

} // namespace hetuCTR
