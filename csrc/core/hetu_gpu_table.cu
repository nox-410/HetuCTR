#include "hetu_gpu_table.h"

#include "common/hip/hip_runtime_api.h"

using namespace hetu;

void HetuGPUTable::pushPull(unsigned long grad, unsigned long dst) {
  generateGradient((embed_t*)grad);

  generateQuery();

  all2allExchangeQuery();

  handleQuery();

  writeBack((embed_t*)dst);
  return;
}
