#include "hetu_gpu_table.h"

#include "common/hip/hip_runtime_api.h"

using namespace hetu;

void HetuGPUTable::pushPull(unsigned long grad, unsigned long dst) {
  generateQuery();
  // Compute shape for query items
  all2allExchangeQuery();
  checkCudaErrors(hipStreamSynchronize(stream_main_));
  return;
}
