#include "hetu_gpu_table.h"

#include "common/hip/hip_runtime_api.h"

using namespace hetu;

void HetuGPUTable::pushPull(unsigned long grad, unsigned long dst) {
  generateGradient((embed_t*)grad);

  all2allGradient();

  generateQuery();

  all2allExchangeQuery();

  handleGradient();

  handleQuery();

  writeBack((embed_t*)dst);
  return;
}
