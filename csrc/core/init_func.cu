#include "hip/hip_runtime.h"
#include "hetu_gpu_table.h"
#include "utils/rendezvous.h"
#include "common/hip/hip_runtime_api.h"

#include <chrono>
#include <thrust/partition.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>

#include <hipcub/hipcub.hpp>

using namespace hetu;

void HetuGPUTable::initializeNCCL(const std::string &ip, const int port) {
  checkCudaErrors(hipSetDevice(device_id_));
  checkCudaErrors(hipStreamCreate(&stream_main_));
  checkCudaErrors(hipStreamCreate(&stream_sub_));
  INFO("Start TCP rendezvous at ", ip, ":", port);
  TCPRendezvous tcp(rank_, nrank_, ip, port);
  ncclUniqueId uid;
  if (rank_ == 0) {
    checkCudaErrors(ncclGetUniqueId(&uid));
  }
  tcp.broadcast(&uid, sizeof(uid));
  INFO("NCCL Connection built successfully");
  checkCudaErrors(ncclCommInitRank(&communicator_, nrank_, uid, rank_));
}

struct _PartitionPrediate {
  const int rank;
  const worker_t *d_root;

  _PartitionPrediate(int _rank, worker_t *_d_root) : rank(_rank), d_root(_d_root) {}

  __device__ bool operator()(index_t idx) const { return d_root[idx]!=rank; }
};

void HetuGPUTable::initializeTable(SArray<worker_t> root_id_arr, SArray<index_t> storage_id_arr) {
  // copy root id array, this indicates which worker holds an embedding.
  checkCudaErrors(hipMalloc(
    &d_root_, sizeof(worker_t) * kEmbeddingIDMax));
  checkCudaErrors(hipMemcpy(
    d_root_, root_id_arr.data(), sizeof(worker_t) * kEmbeddingIDMax, hipMemcpyHostToDevice));

  // Prepare keys and values for HashTable
  // key : reordered storage index, non-local embedding first
  // value : memory offset from 0 to kStorageMax
  thrust::device_vector<index_t> key(kStorageMax), value(kStorageMax);
  thrust::sequence(value.begin(), value.end());
  checkCudaErrors(hipMemcpy(
    key.data().get(), storage_id_arr.data(), sizeof(index_t) * kStorageMax, hipMemcpyHostToDevice));
  // reorder key with Predicate
  auto partition_point = thrust::stable_partition(key.begin(), key.end(), _PartitionPrediate(rank_, d_root_));
  hash_table_.insert(key.data().get(), value.data().get(), kStorageMax, stream_main_);

  // We now know how many non-local embeddings we have, allocate gradients and updates memory for them
  // Do not allocate gradients and updates for local embeddings.
  kNonLocalStorageMax = partition_point - key.begin();
  checkCudaErrors(hipMalloc(
    &d_updates_, sizeof(version_t) * kNonLocalStorageMax));
  checkCudaErrors(hipMalloc(
    &d_version_, sizeof(version_t) * kStorageMax));
  checkCudaErrors(hipMalloc(
    &d_embedding_, sizeof(embed_t) * kStorageMax * kEmbeddingWidth));
  checkCudaErrors(hipMalloc(
    &d_gradient_, sizeof(embed_t) * kNonLocalStorageMax * kEmbeddingWidth));

  // Set Gradients and Updates to zero
  checkCudaErrors(hipMemset(
    d_gradient_, 0, sizeof(embed_t) * kNonLocalStorageMax * kEmbeddingWidth));
  checkCudaErrors(hipMemset(
    d_updates_, 0, sizeof(version_t) * kNonLocalStorageMax));

  // Initialize version, set local version to 1, set non-local version to invalid
  auto v_ptr = thrust::device_ptr<version_t>(d_version_);
  thrust::fill(v_ptr, v_ptr + kNonLocalStorageMax, kInvalidVersion);
  thrust::fill(v_ptr + kNonLocalStorageMax, v_ptr + kStorageMax, 1);
  checkCudaErrors(hipStreamSynchronize(stream_main_));
}

HetuGPUTable::HetuGPUTable(
  const worker_t rank,
  const worker_t nrank,
  const worker_t device_id,
  const std::string &ip,
  const int port,
  const size_t embedding_length,
  const size_t embedding_width,
  const version_t pull_bound,
  const version_t push_bound,
  SArray<worker_t> root_id_arr,
  SArray<index_t> storage_id_arr,
  const Initializer &init,
  const int verbose
) :
  rank_(rank),
  nrank_(nrank),
  device_id_(device_id),
  kEmbeddingIDMax(embedding_length),
  kEmbeddingWidth(embedding_width),
  kStorageMax(storage_id_arr.size()),
  pull_bound_(pull_bound),
  push_bound_(push_bound),
  hash_table_(kStorageMax, 0),
  verbose_(verbose)
{
  // Check device id
  int num_gpus = 0;
  checkCudaErrors(hipGetDeviceCount(&num_gpus));
  CHECK(device_id < num_gpus) << "InCorrect device number.";

  // Setup NCCL
  initializeNCCL(ip, port);

  // Setup Embedding Table
  initializeTable(root_id_arr, storage_id_arr);

  // Initialize Embedding Table with initializer
  unsigned int seed = 0;
  seed = std::chrono::system_clock::now().time_since_epoch().count();
  initialize(d_embedding_, kEmbeddingIDMax * kEmbeddingWidth, init, false, seed);
  INFO("Table Init Successfully");

  // Initialize preprocess data and auxillary memory
  createPreprocessData(cur_batch_, batch_size_reserved_, nrank_);
  createPreprocessData(prev_batch_, batch_size_reserved_, nrank_);
  allocateAuxillaryMemory(batch_size_reserved_);
}

void HetuGPUTable::allocateAuxillaryMemory(size_t batch_size) {
  freeAuxillaryMemory();
  size_t temp_bytes_max = 1, temp_bytes;

  // Check how much temp memory cub want to use
  index_t *ptr;
  checkCudaErrors(hipcub::DeviceRadixSort::SortPairs(
    nullptr, temp_bytes, ptr, ptr, ptr, ptr, batch_size));
  temp_bytes_max = std::max(temp_bytes, temp_bytes_max);
  checkCudaErrors(hipcub::DeviceRunLengthEncode::Encode(
    nullptr, temp_bytes, ptr, ptr, ptr, ptr, batch_size));
  temp_bytes_max = std::max(temp_bytes, temp_bytes_max);

  INFO("Allocate Temp Memory --- ", temp_bytes_max, " bytes");
  checkCudaErrors(hipMalloc(&d_temp_, temp_bytes_max));
  batch_size_reserved_ = batch_size;
  temp_bytes_ = temp_bytes_max;

  // We need to allocate nrank * batch_size so that it will be enough for all-to-all query
  size_t batch_limit = batch_size * nrank_;
  checkCudaErrors(hipMalloc(
    &d_query_idx_, batch_limit * sizeof(index_t)));
  checkCudaErrors(hipMalloc(
    &d_query_gradient_idx_, batch_limit * sizeof(index_t)));
  checkCudaErrors(hipMalloc(
    &d_query_version_, batch_limit * sizeof(version_t)));
  checkCudaErrors(hipMalloc(
    &d_query_updates_, batch_limit * sizeof(version_t)));
  checkCudaErrors(hipMalloc(
    &d_query_val_, batch_limit * sizeof(embed_t) * kEmbeddingWidth));
}

void HetuGPUTable::freeAuxillaryMemory() {
  checkCudaErrors(hipFree(d_temp_));
  checkCudaErrors(hipFree(d_query_idx_));
  checkCudaErrors(hipFree(d_query_gradient_idx_));
  checkCudaErrors(hipFree(d_query_version_));
  checkCudaErrors(hipFree(d_query_updates_));
  checkCudaErrors(hipFree(d_query_val_));
}

HetuGPUTable::~HetuGPUTable() {
  checkCudaErrors(ncclCommDestroy(communicator_));
  checkCudaErrors(hipStreamDestroy(stream_main_));
  checkCudaErrors(hipStreamDestroy(stream_sub_));
  checkCudaErrors(hipFree(d_embedding_));
  checkCudaErrors(hipFree(d_version_));
  checkCudaErrors(hipFree(d_gradient_));
  checkCudaErrors(hipFree(d_updates_));
  checkCudaErrors(hipFree(d_root_));
  freePreprocessData(cur_batch_);
  freePreprocessData(prev_batch_);
  freeAuxillaryMemory();
}
