#include "hetu_gpu_table.h"
#include "utils/rendezvous.h"
#include "common/hip/hip_runtime_api.h"

#include <chrono>
#include <thrust/partition.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>

using namespace hetu;

void HetuGPUTable::initializeNCCL(const std::string &ip, const int port) {
  checkCudaErrors(hipSetDevice(rank_));
  checkCudaErrors(hipStreamCreate(&stream_main_));
  checkCudaErrors(hipStreamCreate(&stream_sub_));
  TCPRendezvous tcp(rank_, nrank_, ip, port);
  ncclUniqueId uid;
  if (rank_ == 0) {
    checkCudaErrors(ncclGetUniqueId(&uid));
  }
  tcp.broadcast(&uid, sizeof(uid));
  checkCudaErrors(ncclCommInitRank(&communicator_, nrank_, uid, rank_));
}

struct _PartitionPrediate {
  const int rank;
  const worker_t *d_root;

  _PartitionPrediate(int _rank, worker_t *_d_root) : rank(_rank), d_root(_d_root) {}

  __device__ bool operator()(index_t idx) const { return d_root[idx]!=rank; }
};

void HetuGPUTable::initializeTable(SArray<worker_t> root_id_arr, SArray<index_t> storage_id_arr) {
  // copy root id array, this indicates which worker holds an embedding.
  checkCudaErrors(hipMalloc(
    &d_root_, sizeof(worker_t) * kEmbeddingIDMax));
  checkCudaErrors(hipMemcpy(
    d_root_, root_id_arr.data(), sizeof(worker_t) * kEmbeddingIDMax, hipMemcpyHostToDevice));

  // Prepare keys and values for HashTable
  // key : reordered storage index, non-local embedding first
  // value : memory offset from 0 to kStorageMax
  thrust::device_vector<index_t> key(kStorageMax), value(kStorageMax);
  thrust::sequence(value.begin(), value.end());
  checkCudaErrors(hipMemcpy(
    key.data().get(), storage_id_arr.data(), sizeof(index_t) * kStorageMax, hipMemcpyHostToDevice));
  // reorder key with Predicate
  auto partition_point = thrust::partition(key.begin(), key.end(), _PartitionPrediate(rank_, d_root_));
  hash_table_.insert(key.data().get(), value.data().get(), kStorageMax, stream_main_);

  // We now know how many non-local embeddings we have, allocate gradients and updates memory for them
  // Do not allocate gradients and updates for local embeddings.
  kNonLocalStorageMax = partition_point - key.begin();
  checkCudaErrors(hipMalloc(
    &d_updates_, sizeof(version_t) * kNonLocalStorageMax));
  checkCudaErrors(hipMalloc(
    &d_version_, sizeof(version_t) * kStorageMax));
  checkCudaErrors(hipMalloc(
    &d_embedding_, sizeof(embed_t) * kStorageMax * kEmbeddingWidth));
  checkCudaErrors(hipMalloc(
    &d_gradient_, sizeof(embed_t) * kNonLocalStorageMax * kEmbeddingWidth));

  // Set Gradients and Updates to zero
  checkCudaErrors(hipMemset(
    d_gradient_, 0, sizeof(embed_t) * kNonLocalStorageMax * kEmbeddingWidth));
  checkCudaErrors(hipMemset(
    d_updates_, 0, sizeof(version_t) * kNonLocalStorageMax));

  // Initialize version, set local version to 1, set non-local version to invalid
  auto v_ptr = thrust::device_ptr<version_t>(d_version_);
  thrust::fill(v_ptr, v_ptr + kNonLocalStorageMax, kInvalidVersion);
  thrust::fill(v_ptr + kNonLocalStorageMax, v_ptr + kStorageMax, 1);
  checkCudaErrors(hipStreamSynchronize(stream_main_));
}

HetuGPUTable::HetuGPUTable(
  const worker_t rank,
  const worker_t nrank,
  const worker_t device_id,
  const std::string &ip,
  const int port,
  const size_t embedding_length,
  const size_t embedding_width,
  const version_t pull_bound,
  const version_t push_bound,
  SArray<worker_t> root_id_arr,
  SArray<index_t> storage_id_arr,
  const Initializer &init
) :
  rank_(rank),
  nrank_(nrank),
  device_id_(device_id),
  kEmbeddingIDMax(embedding_length),
  kEmbeddingWidth(embedding_width),
  kStorageMax(storage_id_arr.size()),
  pull_bound_(pull_bound),
  push_bound_(push_bound),
  hash_table_(kStorageMax, 0)
{
  initializeNCCL(ip, port);
  initializeTable(root_id_arr, storage_id_arr);
  unsigned int seed = 0;
  seed = std::chrono::system_clock::now().time_since_epoch().count();
  initialize(d_embedding_, kEmbeddingIDMax * kEmbeddingWidth, init, false, seed);
}

HetuGPUTable::~HetuGPUTable() {
  checkCudaErrors(ncclCommDestroy(communicator_));
  checkCudaErrors(hipStreamDestroy(stream_main_));
  checkCudaErrors(hipStreamDestroy(stream_sub_));
  checkCudaErrors(hipFree(d_embedding_));
  checkCudaErrors(hipFree(d_version_));
  checkCudaErrors(hipFree(d_gradient_));
  checkCudaErrors(hipFree(d_updates_));
  checkCudaErrors(hipFree(d_root_));
}
