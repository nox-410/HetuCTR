#include "hip/hip_runtime.h"
#include "hetu_gpu_table.h"

#include <cmath>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "common/hip/hip_runtime_api.h"

namespace hetuCTR {

// This computes keys as <root_id, embedding_id>
__global__ void generate_sort_kv_kernel(HetuTable *tbl) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < tbl->cur_batch_.batch_size) {
    index_t embedding_idx = tbl->cur_batch_.d_idx[id];
    assert(embedding_idx < tbl->kEmbeddingIDMax);
    worker_t r = tbl->d_root_[embedding_idx];
    tbl->cur_batch_.d_idx_map[id] = embedding_idx + tbl->kEmbeddingIDMax * r;
    tbl->cur_batch_.d_offset[id] = id;
  }
}

__global__ void block_cvt_offset_to_shape_kernel(size_t *dst) {
  size_t id = threadIdx.x;
  size_t n = blockDim.x;
  extern __shared__ size_t shm[];
  size_t val = dst[id];
  shm[id] = val;
  __syncthreads();
  size_t val_nxt = id == n - 1 ? val : shm[id + 1];
  assert(val_nxt >= val);
  dst[id] = val_nxt - val;
}

__global__ void write_sort_result_kernel(HetuTable *tbl) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < tbl->cur_batch_.batch_size) {
    index_t arg = tbl->cur_batch_.d_sorted_arg[id];
    index_t embedding_idx = tbl->cur_batch_.d_idx[arg];
    tbl->cur_batch_.d_offset[id] = embedding_idx;
  }
}

// This will compute cur_batch_.d_idx_map
// cur_batch_.d_root cur_batch_.u_shape
__global__ void preprocess_batch_data_kernel(HetuTable *tbl) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  size_t n = tbl->cur_batch_.unique_size;
  if (id < n) {
    index_t uid = tbl->cur_batch_.d_unique_idx[id];
    int r = tbl->d_root_[uid], r_prev;
    tbl->cur_batch_.d_root[id] = r;
    auto iter = tbl->table_->find(uid);
    if (iter == tbl->table_->end()) {
      tbl->cur_batch_.d_offset[id] = kInvalidIndex;
    } else {
      tbl->cur_batch_.d_offset[id] = iter->second;
    }
    if (id == 0) r_prev = -1;
    else r_prev = tbl->d_root_[tbl->cur_batch_.d_unique_idx[id - 1]];
    for (int i = r_prev + 1; i <= r; i++) {
      tbl->cur_batch_.u_shape[i] = id;
    }
    if (id == n - 1) {
      for (int i = r + 1; i <= tbl->nrank_; i++) {
        tbl->cur_batch_.u_shape[i] = n;
      }
    }

    // This computes where we can find the unique index from the original index
    index_t idx_start, idx_end;
    idx_start = tbl->cur_batch_.d_run_length[id];
    idx_end = tbl->cur_batch_.d_run_length[id + 1];
    for (index_t i = idx_start; i < idx_end; i++) {
      index_t arg = tbl->cur_batch_.d_sorted_arg[i];
      tbl->cur_batch_.d_idx_map[arg] = id;
    }
  }
}

void HetuTable::preprocessIndex(index_t *data, size_t batch_size) {
  if (batch_size == 0)
    checkCudaErrors(hipMemsetAsync(
      cur_batch_.u_shape, 0, sizeof(size_t) * (nrank_ + 1), stream_main_));

  // Copy batch embedding index data into Device
  checkCudaErrors(hipMemcpyAsync(
    cur_batch_.d_idx, data, sizeof(index_t) * batch_size, hipMemcpyDefault, stream_main_));

  // use unused memory here to store temp sort keys
  generate_sort_kv_kernel<<<DIM_GRID(batch_size), DIM_BLOCK, 0, stream_main_>>>(d_this);
  // we don't need to sort all the bits when using radix sort.
  // using end_bit smaller than 64 can yield corresponding performance improvement
  int end_bit = std::ceil(std::log2(kEmbeddingIDMax * nrank_));
  // store temp unused temp result in d_offset
  checkCudaErrors(hipcub::DeviceRadixSort::SortPairs(
    d_temp_, temp_bytes_, cur_batch_.d_idx_map, cur_batch_.d_unique_idx, cur_batch_.d_offset, cur_batch_.d_sorted_arg,
    batch_size, 0, end_bit, stream_main_));

  // After argsort write value to d_offset (temp, modify in next step)
  write_sort_result_kernel<<<DIM_GRID(batch_size), DIM_BLOCK, 0, stream_main_>>>(d_this);

  // perform unique operation, store total number of unique embedding items;
  checkCudaErrors(hipcub::DeviceRunLengthEncode::Encode(
    d_temp_, temp_bytes_, cur_batch_.d_offset, cur_batch_.d_unique_idx, cur_batch_.d_run_length,
    &(d_this->cur_batch_.unique_size), batch_size, stream_main_));

  checkCudaErrors(hipMemcpyAsync(&cur_batch_.unique_size, &(d_this->cur_batch_.unique_size),
    sizeof(size_t), hipMemcpyDeviceToHost, stream_main_));

  // Store the predix sum of length, this will be used in gradient reduction
  // although we should compute [0, unique_size), but we don't want to sync here
  checkCudaErrors(hipcub::DeviceScan::ExclusiveSum(d_temp_, temp_bytes_,
    cur_batch_.d_run_length, cur_batch_.d_run_length, cur_batch_.batch_size + 1, stream_main_));

  // Computes other preprocess data
  preprocess_batch_data_kernel<<<DIM_GRID(cur_batch_.batch_size), DIM_BLOCK, 0, stream_main_>>>(d_this);

  // convert offset to shape
  block_cvt_offset_to_shape_kernel<<<1, nrank_ + 1,
    sizeof(size_t) * (nrank_ + 1), stream_main_>>>(cur_batch_.u_shape);

  // exchange shape with other workers
  all2allExchangeShape(cur_batch_.u_shape, cur_batch_.u_shape_exchanged);

  checkCudaErrors(hipMemcpyAsync(cur_batch_.h_shape, cur_batch_.u_shape,
    sizeof(size_t) * (nrank_ + 1), hipMemcpyDeviceToHost, stream_main_));
  checkCudaErrors(hipMemcpyAsync(cur_batch_.h_shape_exchanged, cur_batch_.u_shape_exchanged,
    sizeof(size_t) * (nrank_ + 1), hipMemcpyDeviceToHost, stream_main_));
}

// figure out all gradients to push
// 1. compute d_need_update_ as 0 or 1
// 2. update d_version_ (stored and root=self)
// 3. update d_updates_ (stored and root!=self)
//
__global__ void decide_update_kernel(HetuTable *tbl) {
  const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < tbl->prev_batch_.unique_size) {
    version_t update_new = tbl->prev_batch_.d_run_length[id + 1] - tbl->prev_batch_.d_run_length[id];
    index_t offset = tbl->prev_batch_.d_offset[id];
    if (tbl->prev_batch_.d_root[id] == tbl->rank_) {
      tbl->d_need_update_[id] = 0;
      tbl->d_version_[offset] += update_new;
    } else if (offset == kInvalidIndex) {
      tbl->d_need_update_[id] = 1;
    } else {
      // assert(offset < tbl->kNonLocalStorageMax);
      version_t update_local = tbl->d_updates_[offset];
      tbl->d_need_update_[id] = update_local + update_new <= tbl->push_bound_ ? 0 : 1;
      tbl->d_updates_[offset] += update_new;
    }
    if (tbl->d_need_update_[id])
      atomicAdd(&tbl->prev_batch_.u_shape[tbl->prev_batch_.d_root[id]], 1);
  }
}

void HetuTable::preprocessGradient() {
  checkCudaErrors(hipMemsetAsync(prev_batch_.u_shape, 0, nrank_ * sizeof(size_t), stream_main_));
  size_t num_unique = prev_batch_.unique_size;
  decide_update_kernel<<<DIM_GRID(num_unique), DIM_BLOCK, 0, stream_main_>>>(d_this);

  // d_update_prefix_[i] stores which index maps to the gradient communication slot i
  checkCudaErrors(hipcub::DeviceScan::ExclusiveSum(d_temp_, temp_bytes_,
    d_need_update_, d_update_prefix_, num_unique, stream_main_));

  all2allExchangeShape(prev_batch_.u_shape, prev_batch_.u_shape_exchanged);

  checkCudaErrors(hipMemcpyAsync(prev_batch_.h_shape, prev_batch_.u_shape,
    sizeof(size_t) * (nrank_ + 1), hipMemcpyDeviceToHost, stream_main_));
  checkCudaErrors(hipMemcpyAsync(prev_batch_.h_shape_exchanged, prev_batch_.u_shape_exchanged,
    sizeof(size_t) * (nrank_ + 1), hipMemcpyDeviceToHost, stream_main_));
}

} // namespace hetuCTR
