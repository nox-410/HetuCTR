#include "hip/hip_runtime.h"
#include "hetu_gpu_table.h"
#include "common/hip/hip_runtime_api.h"
#include <hipcub/hipcub.hpp>

using namespace hetu;

// aggregate all the gradients into storage
__global__ void table_update_kernel(HetuGPUTable *tbl, embed_t *grad) {
  const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t width = tbl->kEmbeddingWidth;
  if (id >= tbl->prev_batch_.unique_size) return;

  bool need_update = tbl->d_need_update_[id];
  index_t offset = tbl->prev_batch_.d_offset[id];

  int update_type;
  if (offset == kInvalidIndex)
    update_type = 0; // not storaged
  else if (offset >= tbl->kNonLocalStorageMax)
    update_type = 1; // storaged, local
  else if (need_update)
    update_type = 2; // storaged, non-local, update
  else update_type = 3; // storaged, non-local, no update

  index_t query_idx = tbl->d_update_prefix_[id];

  embed_t *dest_grad = tbl->d_gradient_ + offset * width;
  embed_t *dest_query = &tbl->d_query_val_[0][query_idx * width];
  embed_t *dest = tbl->d_embedding_ + offset * width;

  index_t l = tbl->prev_batch_.d_run_length[id], r = tbl->prev_batch_.d_run_length[id + 1];

  if (need_update) {
    version_t update_count = r - l;
    if (update_type == 2) {
      update_count = tbl->d_updates_[offset];
      tbl->d_updates_[offset] = 0;
    }
    tbl->d_query_gradient_idx_[0][query_idx] = tbl->prev_batch_.d_unique_idx[id];
    tbl->d_query_updates_[0][query_idx] = update_count;
  }

  for (size_t i = 0; i < width; i++) {
    embed_t sum = 0;
    for (index_t j = l; j < r; j++) {
      index_t grad_offset = tbl->prev_batch_.d_sorted_arg[j];
      sum += grad[grad_offset * width + i];
    }
    if (update_type != 0) dest[i] += sum;
    else dest_query[i] = sum;

    if (update_type >= 2) dest_grad[i] += sum;
    if (update_type == 2) {
      dest_query[i] = dest_grad[i];
      dest_grad[i] = 0;
    }
  }
}

void HetuGPUTable::generateGradient(embed_t *grad) {
  size_t num_unique = prev_batch_.unique_size;

  table_update_kernel<<<DIM_GRID(num_unique), DIM_BLOCK, 0, stream_main_>>>(this, grad);
}

__global__ void LookUpVersion(HetuGPUTable *tbl) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < tbl->cur_batch_.unique_size) {
    index_t idx = tbl->cur_batch_.d_offset[id];
    if (idx >= 0) tbl->d_query_version_[0][id] = tbl->d_version_[idx];
    else tbl->d_query_version_[0][id] = kInvalidVersion;
  }
}

void HetuGPUTable::generateQuery() {
  // generate local version for each embedding lookup
  LookUpVersion<<<DIM_GRID(cur_batch_.unique_size), DIM_BLOCK, 0, stream_main_>>>(this);
  // Copy index to query buffer
  checkCudaErrors(hipMemcpyAsync(
    d_query_idx_[0], cur_batch_.d_unique_idx, cur_batch_.unique_size * sizeof(index_t), hipMemcpyDeviceToDevice, stream_main_));
}
