#include "hip/hip_runtime.h"
#include "hetu_gpu_table.h"
#include "common/hip/hip_runtime_api.h"

using namespace hetu;

__global__ void LookUpVersion(version_t* dst, const version_t* src, const index_t* offset, const int len) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len) {
    index_t idx = offset[id];
    if (idx >= 0) dst[id] = src[idx];
    else dst[id] = kInvalidVersion;
  }
}

__global__ void LookUpVersion(HetuGPUTable *tbl) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < tbl->cur_batch_.unique_size) {
    index_t idx = tbl->cur_batch_.d_offset[id];
    if (idx >= 0) tbl->d_query_version_[0][id] = tbl->d_version_[idx];
    else tbl->d_query_version_[0][id] = kInvalidVersion;
  }
}

void HetuGPUTable::generateQuery() {
  // generate local version for each embedding lookup
  LookUpVersion<<<DIM_GRID(cur_batch_.unique_size), DIM_BLOCK, 0, stream_main_>>>(this);
  // Copy index to query buffer
  checkCudaErrors(hipMemcpyAsync(
    d_query_idx_[0], cur_batch_.d_unique_idx, cur_batch_.unique_size * sizeof(index_t), hipMemcpyDeviceToDevice, stream_main_));
}

__global__ void computeReturnOutdated(HetuGPUTable *g, size_t len) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len) {
    g->d_embedding_[id] = 0;
  }
}

void HetuGPUTable::handleQuery() {
  size_t num_rcvd = 0;
  for (int i = 0; i < nrank_; i++) num_rcvd += cur_batch_.u_shape_exchanged[i];
  INFO(num_rcvd, " received embedding index to handle.");
  computeReturnOutdated<<<1, 256, 0, stream_main_>>>(this, 256);
}
