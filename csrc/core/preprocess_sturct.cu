#include "preprocess_struct.h"

#include <cassert>
#include <hip/hip_runtime.h>
#include "common/hip/hip_runtime_api.h"

namespace hetuCTR{

void createPreprocessData(PreprocessData &pdata, size_t batch_size, size_t nrank) {
  assert(batch_size > 0);
  pdata.batch_size = 0;
  pdata.unique_size = 0;
  pdata.allocate_size = batch_size;
  checkCudaErrors(hipMalloc(
    &pdata.d_idx, sizeof(index_t) * batch_size));
  checkCudaErrors(hipMalloc(
    &pdata.d_unique_idx, sizeof(index_t) * batch_size));
  checkCudaErrors(hipMalloc(
    &pdata.d_idx_map, sizeof(index_t) * batch_size));
  checkCudaErrors(hipMalloc(
    &pdata.d_offset, sizeof(index_t) * batch_size));
  checkCudaErrors(hipMalloc(
    &pdata.d_root, sizeof(worker_t) * batch_size));
  checkCudaErrors(hipMalloc(
    &pdata.d_run_length, sizeof(index_t) * (batch_size + 1)));
  checkCudaErrors(hipMalloc(
    &pdata.d_sorted_arg, sizeof(index_t) * batch_size));
  checkCudaErrors(hipMalloc(
    &pdata.u_shape, sizeof(size_t) * (nrank + 1)));
  checkCudaErrors(hipMalloc(
    &pdata.u_shape_exchanged, sizeof(size_t) * (nrank + 1)));
  checkCudaErrors(hipHostMalloc(
    &pdata.h_shape, sizeof(size_t) * (nrank + 1)));
  checkCudaErrors(hipHostMalloc(
    &pdata.h_shape_exchanged, sizeof(size_t) * (nrank + 1)));
}

void freePreprocessData(PreprocessData &pdata) {
  checkCudaErrors(hipFree(pdata.d_idx));
  checkCudaErrors(hipFree(pdata.d_unique_idx));
  checkCudaErrors(hipFree(pdata.d_idx_map));
  checkCudaErrors(hipFree(pdata.d_offset));
  checkCudaErrors(hipFree(pdata.d_root));
  checkCudaErrors(hipFree(pdata.d_run_length));
  checkCudaErrors(hipFree(pdata.d_sorted_arg));
  checkCudaErrors(hipFree(pdata.u_shape));
  checkCudaErrors(hipFree(pdata.u_shape_exchanged));
  checkCudaErrors(hipHostFree(pdata.h_shape));
  checkCudaErrors(hipHostFree(pdata.h_shape_exchanged));
}

} // namespace hetuCTR
